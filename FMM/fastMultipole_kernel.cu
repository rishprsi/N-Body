#include "hip/hip_runtime.h"
/*
   Copyright 2023 Hsin-Hung Wu

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <stdio.h>
#include <math.h>
#include "constants.h"
#include "fastMultipole_kernel.cuh"

// Helper device functions for complex arithmetic
__device__ Complex complexAdd(Complex a, Complex b) {
    return {a.real + b.real, a.imag + b.imag};
}

__device__ Complex complexMul(Complex a, Complex b) {
    return {a.real * b.real - a.imag * b.imag, a.real * b.imag + a.imag * b.real};
}

__device__ Complex complexScale(Complex a, double scale) {
    return {a.real * scale, a.imag * scale};
}

// Get quadrant for a position relative to a center
__device__ int getQuadrant(Vector position, Vector center) {
    int quadrant = 0;
    if (position.x >= center.x) {
        if (position.y >= center.y) {
            quadrant = 0; // Top-right
        } else {
            quadrant = 3; // Bottom-right
        }
    } else {
        if (position.y >= center.y) {
            quadrant = 1; // Top-left
        } else {
            quadrant = 2; // Bottom-left
        }
    }
    return quadrant;
}

// Custom atomic operations for double precision values
__device__ double atomicMin(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(min(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    
    return __longlong_as_double(old);
}

__device__ double atomicMax(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(max(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    
    return __longlong_as_double(old);
}

// Compute bounding box for all bodies
__global__ void ComputeBoundingBoxKernel(Body *bodies, Cell *cells, int *mutex, int nBodies) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBodies) return;
    
    Body body = bodies[idx];
    
    // Update root cell bounds using atomic operations
    if (idx == 0) {
        // Initialize root cell
        cells[0].center = {0, 0};
        cells[0].size = NBODY_WIDTH;
        cells[0].parent = -1;
        cells[0].children[0] = cells[0].children[1] = cells[0].children[2] = cells[0].children[3] = -1;
        cells[0].bodyStart = 0;
        cells[0].bodyCount = nBodies;
        cells[0].isLeaf = true;
        cells[0].totalMass = 0.0;
        
        // Initialize multipole and local expansions
        for (int i = 0; i < P; i++) {
            cells[0].multipole[i] = {0, 0};
            cells[0].local[i] = {0, 0};
        }
        
        // Initialize bounds to extreme values
        cells[0].minBound = {INFINITY, INFINITY};
        cells[0].maxBound = {-INFINITY, -INFINITY};
    }
    
    // Ensure root cell is initialized before updating bounds
    __syncthreads();
    
    // Update bounding box using atomic operations (following BH style)
    atomicMin(&cells[0].minBound.x, body.position.x);
    atomicMin(&cells[0].minBound.y, body.position.y);
    atomicMax(&cells[0].maxBound.x, body.position.x);
    atomicMax(&cells[0].maxBound.y, body.position.y);
}

// Build the quadtree
__global__ void BuildTreeKernel(Body *bodies, Cell *cells, int *cellCount, int *sortedIndex, int *mutex, int nBodies, int maxDepth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBodies) return;
    
    // Initialize sorted index
    sortedIndex[idx] = idx;
    
    // Create root cell if this is the first thread
    if (idx == 0) {
        int rootIdx = atomicAdd(cellCount, 1);
        cells[rootIdx].center = {0, 0};
        cells[rootIdx].size = NBODY_WIDTH;
        cells[rootIdx].parent = -1;
        cells[rootIdx].bodyStart = 0;
        cells[rootIdx].bodyCount = nBodies;
        cells[rootIdx].isLeaf = (nBodies <= MAX_PARTICLES_PER_LEAF);
        
        for (int i = 0; i < 4; i++) {
            cells[rootIdx].children[i] = -1;
        }
        
        // Initialize multipole and local expansions
        for (int i = 0; i < P; i++) {
            cells[rootIdx].multipole[i] = {0, 0};
            cells[rootIdx].local[i] = {0, 0};
        }
    }
    
    __syncthreads();
    
    // Insert body into the tree
    Body body = bodies[idx];
    int cellIdx = 0; // Start at root
    int depth = 0;
    
    while (depth < maxDepth && !cells[cellIdx].isLeaf) {
        // Determine which quadrant the body belongs to
        int quadrant = getQuadrant(body.position, cells[cellIdx].center);
        
        // Check if child cell exists
        if (cells[cellIdx].children[quadrant] == -1) {
            // Create new child cell
            int newCellIdx = atomicAdd(cellCount, 1);
            cells[cellIdx].children[quadrant] = newCellIdx;
            
            // Calculate new center and size
            double halfSize = cells[cellIdx].size / 2.0;
            Vector center = cells[cellIdx].center;
            
            if (quadrant == 0) { // Top-right
                center.x += halfSize / 2.0;
                center.y += halfSize / 2.0;
            } else if (quadrant == 1) { // Top-left
                center.x -= halfSize / 2.0;
                center.y += halfSize / 2.0;
            } else if (quadrant == 2) { // Bottom-left
                center.x -= halfSize / 2.0;
                center.y -= halfSize / 2.0;
            } else { // Bottom-right
                center.x += halfSize / 2.0;
                center.y -= halfSize / 2.0;
            }
            
            // Initialize new cell
            cells[newCellIdx].center = center;
            cells[newCellIdx].size = halfSize;
            cells[newCellIdx].parent = cellIdx;
            cells[newCellIdx].bodyStart = 0;
            cells[newCellIdx].bodyCount = 0;
            cells[newCellIdx].isLeaf = true;
            
            for (int i = 0; i < 4; i++) {
                cells[newCellIdx].children[i] = -1;
            }
            
            // Initialize multipole and local expansions
            for (int i = 0; i < P; i++) {
                cells[newCellIdx].multipole[i] = {0, 0};
                cells[newCellIdx].local[i] = {0, 0};
            }
        }
        
        // Move to child cell
        cellIdx = cells[cellIdx].children[quadrant];
        depth++;
    }
    
    // Add body to leaf cell
    int bodyIdx = atomicAdd(&cells[cellIdx].bodyCount, 1);
    if (bodyIdx == 0) {
        cells[cellIdx].bodyStart = idx;
    }
    
    // If leaf is full, mark it as non-leaf for next iteration
    if (cells[cellIdx].bodyCount > MAX_PARTICLES_PER_LEAF && depth < maxDepth) {
        cells[cellIdx].isLeaf = false;
    }
}

// Compute multipole expansions for leaf cells
__global__ void ComputeMultipolesKernel(Body *bodies, Cell *cells, int *sortedIndex, int nCells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nCells) return;
    
    Cell cell = cells[idx];
    
    // Only process leaf cells
    if (cell.isLeaf && cell.bodyCount > 0) {
        // Compute multipole expansion for this cell
        computeMultipoleExpansion(bodies, cell.bodyStart, cell.bodyCount, cells[idx].multipole, cell.center);
    }
}

// Compute multipole expansion for a group of bodies
__device__ void computeMultipoleExpansion(Body *bodies, int start, int count, Complex *multipole, Vector center) {
    // Initialize multipole coefficients
    for (int i = 0; i < P; i++) {
        multipole[i] = {0, 0};
    }
    
    // Monopole term (p=0) is just the total mass
    double totalMass = 0;
    for (int i = 0; i < count; i++) {
        Body body = bodies[start + i];
        totalMass += body.mass;
    }
    multipole[0] = {totalMass, 0};
    
    // Higher order terms
    for (int i = 0; i < count; i++) {
        Body body = bodies[start + i];
        
        // Convert to complex coordinates relative to cell center
        double dx = body.position.x - center.x;
        double dy = body.position.y - center.y;
        Complex z = {dx, dy};
        
        // Compute powers of z
        Complex zpow = {1, 0}; // z^0 = 1
        
        for (int p = 1; p < P; p++) {
            // z^p = z^(p-1) * z
            zpow = complexMul(zpow, z);
            
            // Add contribution to multipole coefficient
            Complex contrib = complexScale(zpow, body.mass);
            multipole[p] = complexAdd(multipole[p], contrib);
        }
    }
}

// Translate multipole expansions from children to parent
__global__ void TranslateMultipolesKernel(Cell *cells, int nCells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nCells) return;
    
    Cell cell = cells[idx];
    
    // Skip leaf cells and the root
    if (cell.isLeaf || cell.parent == -1) return;
    
    // Translate multipole expansion to parent
    int parentIdx = cell.parent;
    translateMultipole(cell.multipole, cells[parentIdx].multipole, cell.center, cells[parentIdx].center);
}

// Translate a multipole expansion from source to target
__device__ void translateMultipole(Complex *source, Complex *target, Vector sourceCenter, Vector targetCenter) {
    // Compute translation vector
    double dx = sourceCenter.x - targetCenter.x;
    double dy = sourceCenter.y - targetCenter.y;
    Complex z0 = {dx, dy};
    
    // For each target multipole coefficient
    for (int p = 0; p < P; p++) {
        Complex sum = {0, 0};
        
        // Combine source multipoles with appropriate binomial coefficients
        for (int k = 0; k <= p; k++) {
            // Binomial coefficient C(p,k)
            int binomial = 1;
            for (int i = 1; i <= k; i++) {
                binomial = binomial * (p - i + 1) / i;
            }
            
            // z0^(p-k) * source[k] * C(p,k)
            Complex zpow = {1, 0}; // z0^0 = 1
            for (int i = 0; i < p-k; i++) {
                zpow = complexMul(zpow, z0);
            }
            
            Complex term = complexMul(zpow, source[k]);
            term = complexScale(term, binomial);
            sum = complexAdd(sum, term);
        }
        
        // Add to target multipole
        target[p] = complexAdd(target[p], sum);
    }
}

// Compute local expansions
__global__ void ComputeLocalExpansionsKernel(Cell *cells, int nCells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nCells) return;
    
    // Start from the root and propagate local expansions downward
    if (idx == 0) { // Root cell
        // Root's local expansion is zero (no external influences)
        for (int i = 0; i < P; i++) {
            cells[0].local[i] = {0, 0};
        }
    }
    
    __syncthreads();
    
    Cell cell = cells[idx];
    
    // Skip leaf cells
    if (cell.isLeaf) return;
    
    // For each child, translate parent's local expansion plus siblings' multipole expansions
    for (int childIdx = 0; childIdx < 4; childIdx++) {
        int child = cell.children[childIdx];
        if (child == -1) continue;
        
        // First, translate parent's local expansion to child
        for (int i = 0; i < P; i++) {
            cells[child].local[i] = cells[idx].local[i];
        }
        
        // Then, for each sibling, translate its multipole expansion to child's local expansion
        for (int siblingIdx = 0; siblingIdx < 4; siblingIdx++) {
            int sibling = cell.children[siblingIdx];
            if (sibling == -1 || sibling == child) continue;
            
            translateMultipoleToLocal(cells[sibling].multipole, cells[child].local, 
                                     cells[sibling].center, cells[child].center);
        }
    }
}

// Translate multipole expansion to local expansion
__device__ void translateMultipoleToLocal(Complex *multipole, Complex *local, Vector multipoleCenter, Vector localCenter) {
    // Compute distance between centers
    double dx = multipoleCenter.x - localCenter.x;
    double dy = multipoleCenter.y - localCenter.y;
    double r2 = dx*dx + dy*dy;
    
    // Skip if centers are too close (MAC criterion)
    if (r2 < THETA * THETA) return;
    
    Complex z0 = {dx, dy};
    double r = sqrt(r2);
    
    // For each local coefficient
    for (int p = 0; p < P; p++) {
        Complex sum = {0, 0};
        
        // Combine multipole coefficients
        for (int k = 0; k < P-p; k++) {
            // Compute (1/z0)^(k+p+1) * multipole[k] / k!
            Complex zpow = {1, 0}; // (1/z0)^0 = 1
            for (int i = 0; i < k+p+1; i++) {
                // Compute 1/z0
                double denom = z0.real*z0.real + z0.imag*z0.imag;
                Complex invz = {z0.real/denom, -z0.imag/denom};
                zpow = complexMul(zpow, invz);
            }
            
            // Factorial k!
            int factorial = 1;
            for (int i = 2; i <= k; i++) {
                factorial *= i;
            }
            
            Complex term = complexMul(zpow, multipole[k]);
            term = complexScale(term, 1.0/factorial);
            sum = complexAdd(sum, term);
        }
        
        // Add to local expansion
        local[p] = complexAdd(local[p], sum);
    }
}

// Evaluate local expansions for all bodies
__global__ void EvaluateLocalExpansionsKernel(Body *bodies, Cell *cells, int *sortedIndex, int nBodies) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBodies) return;
    
    Body body = bodies[idx];
    
    // Find the leaf cell containing this body
    int cellIdx = 0; // Start at root
    while (!cells[cellIdx].isLeaf) {
        int quadrant = getQuadrant(body.position, cells[cellIdx].center);
        cellIdx = cells[cellIdx].children[quadrant];
        if (cellIdx == -1) break; // Error case
    }
    
    if (cellIdx == -1) return; // Error case
    
    // Evaluate local expansion at body position
    Vector force = {0, 0};
    evaluateLocalExpansion(cells[cellIdx].local, cells[cellIdx].center, body.position, &force);
    
    // Store force for later use in force computation
    bodies[idx].acceleration = force;
}

// Evaluate local expansion at a position
__device__ void evaluateLocalExpansion(Complex *local, Vector center, Vector position, Vector *force) {
    // Convert to complex coordinates relative to cell center
    double dx = position.x - center.x;
    double dy = position.y - center.y;
    Complex z = {dx, dy};
    
    // Evaluate local expansion
    Complex potential = {0, 0};
    Complex field = {0, 0};
    
    // Compute powers of z
    Complex zpow = {1, 0}; // z^0 = 1
    
    for (int p = 0; p < P; p++) {
        // Add term to potential: local[p] * z^p
        Complex term = complexMul(local[p], zpow);
        potential = complexAdd(potential, term);
        
        // Add term to field: p * local[p] * z^(p-1)
        if (p > 0) {
            Complex fieldTerm = complexMul(local[p], zpow);
            fieldTerm = complexScale(fieldTerm, p);
            field = complexAdd(field, fieldTerm);
        }
        
        // z^(p+1) = z^p * z
        zpow = complexMul(zpow, z);
    }
    
    // Convert complex field to force vector (F = -grad(potential))
    force->x = -field.real * GRAVITY;
    force->y = -field.imag * GRAVITY;
}

// Direct evaluation for nearby particles
__global__ void DirectEvaluationKernel(Body *bodies, Cell *cells, int *sortedIndex, int nBodies) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBodies) return;
    
    Body body1 = bodies[idx];
    Vector force = {0, 0};
    
    // Find the leaf cell containing this body
    int cellIdx = 0; // Start at root
    while (!cells[cellIdx].isLeaf) {
        int quadrant = getQuadrant(body1.position, cells[cellIdx].center);
        cellIdx = cells[cellIdx].children[quadrant];
        if (cellIdx == -1) break; // Error case
    }
    
    if (cellIdx == -1) return; // Error case
    
    // Compute direct interactions with bodies in the same leaf cell
    for (int i = 0; i < cells[cellIdx].bodyCount; i++) {
        int otherIdx = cells[cellIdx].bodyStart + i;
        if (otherIdx == idx) continue; // Skip self
        
        Body body2 = bodies[otherIdx];
        Vector directForce = {0, 0};
        computeDirectForce(body1, body2, &directForce);
        
        force.x += directForce.x;
        force.y += directForce.y;
    }
    
    // Add to acceleration from local expansion
    bodies[idx].acceleration.x += force.x;
    bodies[idx].acceleration.y += force.y;
}

// Compute direct force between two bodies
__device__ void computeDirectForce(Body body1, Body body2, Vector *force) {
    double dx = body2.position.x - body1.position.x;
    double dy = body2.position.y - body1.position.y;
    double r2 = dx*dx + dy*dy;
    
    // Add softening to prevent numerical instability
    r2 = fmax(r2, E * E);
    
    double r = sqrt(r2);
    double f = GRAVITY * body1.mass * body2.mass / r2;
    
    force->x = f * dx / r;
    force->y = f * dy / r;
}

// Final force computation and integration
__global__ void ComputeForcesAndUpdateKernel(Body *bodies, int nBodies) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBodies) return;
    
    Body body = bodies[idx];
    
    // Skip non-dynamic bodies
    if (!body.isDynamic) return;
    
    // Update velocity using acceleration
    body.velocity.x += body.acceleration.x * DT / body.mass;
    body.velocity.y += body.acceleration.y * DT / body.mass;
    
    // Update position using velocity
    body.position.x += body.velocity.x * DT;
    body.position.y += body.velocity.y * DT;
    
    // Reset acceleration for next iteration
    body.acceleration = {0, 0};
    
    // Write back to global memory
    bodies[idx] = body;
}

// Add ResetMutexKernel implementation
__global__ void ResetMutexKernel(int *mutex, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        mutex[idx] = 0;
    }
}

// Reset cells kernel - follows BH's ResetKernel pattern
__global__ void ResetCellsKernel(Cell *cells, int *mutex, int nCells, int nBodies) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < nCells) {
        cells[idx].isLeaf = true;
        cells[idx].parent = -1;
        cells[idx].children[0] = cells[idx].children[1] = cells[idx].children[2] = cells[idx].children[3] = -1;
        cells[idx].bodyStart = -1;
        cells[idx].bodyCount = 0;
        cells[idx].totalMass = 0.0;
        cells[idx].minBound = {INFINITY, INFINITY};
        cells[idx].maxBound = {-INFINITY, -INFINITY};
        
        // Initialize multipole and local expansions
        for (int i = 0; i < P; i++) {
            cells[idx].multipole[i] = {0, 0};
            cells[idx].local[i] = {0, 0};
        }
        
        mutex[idx] = 0;
    }
    
    // Set up root node to handle all bodies (like BH implementation)
    if (idx == 0) {
        cells[idx].bodyStart = 0;
        cells[idx].bodyCount = nBodies;
    }
} 