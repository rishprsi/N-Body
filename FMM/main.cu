#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include "constants.h"
#include "fastMultipoleCuda.cuh"
#include <fstream>
#include <hip/hip_runtime.h>
#include "err.h"

cv::VideoWriter video("nbody.avi", cv::VideoWriter::fourcc('M', 'J', 'P', 'G'), 30, cv::Size(WINDOW_WIDTH, WINDOW_HEIGHT));

Vector scaleToWindow(Vector pos)
{

    double scaleX = WINDOW_HEIGHT / NBODY_HEIGHT;
    double scaleY = WINDOW_WIDTH / NBODY_WIDTH;
    return {(pos.x - 0) * scaleX + WINDOW_WIDTH / 2, (pos.y - 0) * scaleY + WINDOW_HEIGHT / 2};
}

// Helper function to store frames
void storeFrame(Body *bodies, int n, int id, int error_check, Body *naive_bodies)
{
    cv::Mat image = cv::Mat::zeros(WINDOW_HEIGHT, WINDOW_WIDTH, CV_8UC3);
    cv::Scalar color; // White color
    int radius;
    
    for (int i = 0; i < n; i++)
    {
        Vector pos = scaleToWindow(bodies[i].position);
        cv::Point center(pos.x, pos.y);
        // std::cout << " " << pos.x <<" "<< pos.y;
        // stars will be red and planets will be white
        if (bodies[i].mass >= HBL)
        {
            color = cv::Scalar(0, 0, 255);
            radius = 5;
        }
        else
        {
            color = cv::Scalar(255, 255, 255);
            radius = 1;
        }
        cv::circle(image, center, radius, color, -1);
    }
    // std::cout << std::endl;
    if (error_check == 1){
        for (int i = 0; i < n; i++)
    {
        Vector pos2 = scaleToWindow(naive_bodies[i].position);
        cv::Point center(pos2.x, pos2.y);
        // std::cout << " " << pos2.x <<" "<< pos2.y;
        // stars will be red and planets will be white
        if (naive_bodies[i].mass >= HBL)
        {
            color = cv::Scalar(0, 0, 255);
            radius = 5;
        }
        else
        {
            color = cv::Scalar(255, 255,0 );
            radius = 1;
        }
        cv::circle(image, center, radius, color, -1);
    }
    // std::cout << std::endl;
    }
    
    video.write(image);
}

// Check command line arguments
bool checkArgs(int nBodies, int sim, int iter) {
    if (nBodies < 1) {
        std::cout << "ERROR: need to have at least 1 body" << std::endl;
        return false;
    }

    if (sim < 0 || sim > 3) {
        std::cout << "ERROR: simulation doesn't exist" << std::endl;
        return false;
    }

    if (iter < 1) {
        std::cout << "ERROR: need to have at least 1 iteration" << std::endl;
        return false;
    }

    return true;
}

void exportPositionsToCSV(Body *bodies, int n, const std::string &filename) {
    std::ofstream outputFile(filename);
    if (!outputFile.is_open()) {
        std::cerr << "Failed to open " << filename << " for writing" << std::endl;
        return;
    }
    
    // Write header
    outputFile << "id,x,y,mass,velocity_x,velocity_y" << std::endl;
    
    // Write body data
    for (int i = 0; i < n; i++) {
        outputFile << i << ","
                  << bodies[i].position.x << ","
                  << bodies[i].position.y << ","
                  << bodies[i].mass << ","
                  << bodies[i].velocity.x << ","
                  << bodies[i].velocity.y << std::endl;
    }
    
    outputFile.close();
}


// Calculates Divergence between Direct sum and Barnes Hut
double getAvgDivergence(Body *bodies, Body *naive_bodies, int numBodies) {
    double totalDivergence = 0.0;
    Body *temp1 = new Body[numBodies];
    Body *temp2 = new Body[numBodies];

    

    for (int i = 0; i < numBodies; ++i) {
        temp1[bodies[i].id] = bodies[i];
        temp2[naive_bodies[i].id] = naive_bodies[i];
    }
    bodies = temp1;
    naive_bodies = temp2;
    
    for (int i = 0; i < numBodies; ++i) {
        double dx = bodies[i].position.x - naive_bodies[i].position.x;
        double dy = bodies[i].position.y - naive_bodies[i].position.y;
        double divergence = sqrt(dx * dx + dy * dy);
        
        totalDivergence += divergence;
    }
    
    return numBodies > 0 ? totalDivergence / numBodies : 0.0;
}

int main(int argc, char **argv) {
    int nBodies = NUM_BODIES;
    int sim = 0;
    int iters = 300;
    int exportFreq = 10;  // Export every 10 frames
    int error_check = 0;
    
    if (argc > 4)
    {
        nBodies = atoi(argv[1]);
        sim = atoi(argv[2]);
        iters = atoi(argv[3]);
        error_check = atoi(argv[4]);
    }else if (argc==4){
        nBodies = atoi(argv[1]);
        sim = atoi(argv[2]);
        iters = atoi(argv[3]);
    }
    
    if (!checkArgs(nBodies, sim, iters))
        return -1;
    
    if (sim == 3)
        nBodies = 5; // Solar system has fixed number of bodies
    
    FastMultipoleCuda *fmm = new FastMultipoleCuda(nBodies, error_check);
    fmm->setup(sim);

    Body *bodies;
    system("mkdir -p output_data");

    fmm->resetTimers();
    fmm->update();
    fmm->readDeviceBodies();
    bodies = fmm->getBodies();

    if (error_check==1){
        fmm->runNaive();
        storeFrame(bodies,nBodies,0,error_check, fmm->readNaiveDeviceBodies());
    }else{
        storeFrame(bodies, nBodies, 0,error_check,{});
    }

    std::string filename = "output_data/positions_0.csv";
    exportPositionsToCSV(bodies, nBodies, filename);

    // Reset timers after the warm-up iteration
    fmm->resetTimers();
    std::cout << "Starting timed iterations (excluding warm-up iteration)..." << std::endl;
    
    // CUDA events for timing entire execution including memory transfers
    hipEvent_t start, stop;
    float executionTimeMs;
  
    for (int i = 0; i < iters; ++i) {

        // Start timing for full execution including memory transfers
        CHECK_CUDA_ERROR(hipEventCreate(&start));
        CHECK_CUDA_ERROR(hipEventCreate(&stop));
        CHECK_CUDA_ERROR(hipEventRecord(start));

        fmm->update();
        fmm->readDeviceBodies();

        // Record end time for full execution
        CHECK_CUDA_ERROR(hipEventRecord(stop));
        CHECK_CUDA_ERROR(hipEventSynchronize(stop));
        CHECK_CUDA_ERROR(hipEventElapsedTime(&executionTimeMs, start, stop));

        // Add the measured time to our total
        fmm->addExecutionTime(executionTimeMs);

        // Clean up CUDA events
        CHECK_CUDA_ERROR(hipEventDestroy(start));
        CHECK_CUDA_ERROR(hipEventDestroy(stop));
        
        // Visualization and data export
        bodies = fmm->getBodies();

        if (error_check==1){
            fmm->runNaive();
            storeFrame(bodies, nBodies, i,error_check, fmm->readNaiveDeviceBodies());
        }else{
            storeFrame(bodies, nBodies, i,error_check, {});
        }

        if (i % exportFreq == 0) {
            std::string filename = "output_data/positions_" + std::to_string(i) + ".csv";
            exportPositionsToCSV(bodies, nBodies, filename);
        }
    }

    if (error_check==1){
        Body *naive_bodies = fmm->readNaiveDeviceBodies();
        double error = getAvgDivergence(bodies,naive_bodies,nBodies);
        double maxDistance = sqrt(NBODY_WIDTH * NBODY_WIDTH + NBODY_HEIGHT * NBODY_HEIGHT);
        std::cout << "Average error rate is: " << error << std::endl;
        std::cout << "Normalized error rate is: " << error/maxDistance << std::endl;
    }

    // Adjust the iteration count in performance metrics to reflect only the timed iterations
    int timedIterations = iters - 1;
    
    fmm->printPerformanceMetrics();
    
    // Export final performance data
    std::ofstream perfOutput("performance_results.csv");
    if (perfOutput.is_open()) {
        perfOutput << "bodies,iterations,total_kernel_time_ms,avg_kernel_time_ms,total_execution_time_ms,avg_execution_time_ms,total_flops,kernel_gflops,effective_gflops" << std::endl;
        double kernelGflops = fmm->getTotalFlops() / (fmm->getTotalKernelTime() * 1e6);
        double effectiveGflops = fmm->getTotalFlops() / (fmm->getTotalExecutionTime() * 1e6);
        
        perfOutput << nBodies << ","
                  << timedIterations << ","
                  << fmm->getTotalKernelTime() << ","
                  << fmm->getAverageKernelTime() << ","
                  << fmm->getTotalExecutionTime() << ","
                  << fmm->getAverageExecutionTime() << ","
                  << fmm->getTotalFlops() << ","
                  << kernelGflops << ","
                  << effectiveGflops << std::endl;
        
        perfOutput.close();
        std::cout << "Performance data written to performance_results.csv" << std::endl;
    } else {
        std::cerr << "ERROR: Could not open performance_results.csv for writing" << std::endl;
    }
    
    video.release();
    delete fmm;
    
    return 0;
} 