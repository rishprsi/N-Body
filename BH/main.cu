#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <opencv2/opencv.hpp>
#include "barnesHutCuda.cuh"
#include "constants.h"
#include "err.h"

cv::VideoWriter video("nbody.avi", cv::VideoWriter::fourcc('M', 'J', 'P', 'G'), 30, cv::Size(WINDOW_WIDTH, WINDOW_HEIGHT));

Vector scaleToWindow(Vector pos)
{

    double scaleX = WINDOW_HEIGHT / NBODY_HEIGHT;
    double scaleY = WINDOW_WIDTH / NBODY_WIDTH;
    return {(pos.x - 0) * scaleX + WINDOW_WIDTH / 2, (pos.y - 0) * scaleY + WINDOW_HEIGHT / 2};
}

void storeFrame(Body *bodies, int n, int id, int error_check, Body *naive_bodies)
{
    cv::Mat image = cv::Mat::zeros(WINDOW_HEIGHT, WINDOW_WIDTH, CV_8UC3);
    cv::Scalar color; // White color
    int radius;
    
    for (int i = 0; i < n; i++)
    {
        Vector pos = scaleToWindow(bodies[i].position);
        cv::Point center(pos.x, pos.y);
        // std::cout << " " << pos.x <<" "<< pos.y;
        // stars will be red and planets will be white
        if (bodies[i].mass >= HBL)
        {
            color = cv::Scalar(0, 0, 255);
            radius = 5;
        }
        else
        {
            color = cv::Scalar(255, 255, 255);
            radius = 1;
        }
        cv::circle(image, center, radius, color, -1);
    }
    // std::cout << std::endl;
    if (error_check == 1){
        for (int i = 0; i < n; i++)
    {
        Vector pos2 = scaleToWindow(naive_bodies[i].position);
        cv::Point center(pos2.x, pos2.y);
        // std::cout << " " << pos2.x <<" "<< pos2.y;
        // stars will be red and planets will be white
        if (naive_bodies[i].mass >= HBL)
        {
            color = cv::Scalar(0, 0, 255);
            radius = 5;
        }
        else
        {
            color = cv::Scalar(255, 255,0 );
            radius = 1;
        }
        cv::circle(image, center, radius, color, -1);
    }
    // std::cout << std::endl;
    }
    
    video.write(image);
}

bool checkArgs(int nBodies, int sim, int iter)
{

    if (nBodies < 1)
    {
        std::cout << "ERROR: need to have at least 1 body" << std::endl;
        return false;
    }

    if (sim < 0 || sim > 3)
    {
        std::cout << "ERROR: simulation doesn't exist" << std::endl;
        return false;
    }

    if (iter < 1)
    {
        std::cout << "ERROR: need to have at least 1 iteration" << std::endl;
        return false;
    }

    return true;
}

// Calculates Divergence between Direct sum and Barnes Hut
double getAvgDivergence(Body *bodies, Body *naive_bodies, int numBodies) {
    double totalDivergence = 0.0;
    Body *temp1 = new Body[numBodies];
    Body *temp2 = new Body[numBodies];

    

    for (int i = 0; i < numBodies; ++i) {
        temp1[bodies[i].id] = bodies[i];
        temp2[naive_bodies[i].id] = naive_bodies[i];
    }
    bodies = temp1;
    naive_bodies = temp2;
    
    for (int i = 0; i < numBodies; ++i) {
        double dx = bodies[i].position.x - naive_bodies[i].position.x;
        double dy = bodies[i].position.y - naive_bodies[i].position.y;
        double divergence = sqrt(dx * dx + dy * dy);
        
        totalDivergence += divergence;
    }
    
    return numBodies > 0 ? totalDivergence / numBodies : 0.0;
}

int main(int argc, char **argv)
{
    hipDeviceSetLimit(hipLimitStackSize, 8192);

    int nBodies = NUM_BODIES;
    int sim = 0;
    int iters = 300;
    int error_check = 0;
    
    if (argc >= 4)
    {
        nBodies = atoi(argv[1]);
        sim = atoi(argv[2]);
        iters = atoi(argv[3]);
        error_check = atoi(argv[4]);
    }else if (argc==3){
        nBodies = atoi(argv[1]);
        sim = atoi(argv[2]);
        iters = atoi(argv[3]);
    }

    if (!checkArgs(nBodies, sim, iters))
        return -1;

    if (sim == 3)
        nBodies = 5;
        
    std::cout << "Running Barnes-Hut simulation with " << nBodies << " bodies for " 
              << iters << " iterations" << std::endl;

    BarnesHutCuda *bh = new BarnesHutCuda(nBodies,error_check);
    bh->setup(sim);
    Body *bodies;
    

    // First iteration (iteration 0) - warm-up run without timing
    bh->resetTimers();
    bh->update();
    bh->readDeviceBodies();
    bodies = bh->getBodies();
    if (error_check==1){
        bh->runNaive();
        storeFrame(bodies,nBodies,0,error_check, bh->readNaiveDeviceBodies());
    }else{
        storeFrame(bodies, nBodies, 0,error_check,{});
    }
    
        
    // Reset timers after the warm-up iteration
    bh->resetTimers();
    std::cout << "Starting timed iterations (excluding warm-up iteration)..." << std::endl;
    
    // CUDA events for timing entire execution including memory transfers
    hipEvent_t start, stop;
    float executionTimeMs;

    
    
    // Start measuring from iteration 1 onwards
    for (int i = 1; i < iters; ++i)
    {
        // Start timing for full execution including memory transfers
        CHECK_CUDA_ERROR(hipEventCreate(&start));
        CHECK_CUDA_ERROR(hipEventCreate(&stop));
        CHECK_CUDA_ERROR(hipEventRecord(start));
        
        // Computational phase
        bh->update();
        
        // Memory transfer phase
        bh->readDeviceBodies();
        
        // Record end time for full execution
        CHECK_CUDA_ERROR(hipEventRecord(stop));
        CHECK_CUDA_ERROR(hipEventSynchronize(stop));
        CHECK_CUDA_ERROR(hipEventElapsedTime(&executionTimeMs, start, stop));
        
        // Add the measured time to our total
        bh->addExecutionTime(executionTimeMs);

        
        
        // Clean up CUDA events
        CHECK_CUDA_ERROR(hipEventDestroy(start));
        CHECK_CUDA_ERROR(hipEventDestroy(stop));
        
        // Visualization
        bodies = bh->getBodies();
        

        if (error_check==1){
            bh->runNaive();
            storeFrame(bodies, nBodies, i,error_check, bh->readNaiveDeviceBodies());
        }else{
            storeFrame(bodies, nBodies, i,error_check, {});
        }
    }

    if (error_check==1){
        Body *naive_bodies = bh->readNaiveDeviceBodies();
        double error = getAvgDivergence(bodies,naive_bodies,nBodies);
        double maxDistance = sqrt(NBODY_WIDTH * NBODY_WIDTH + NBODY_HEIGHT * NBODY_HEIGHT);
        std::cout << "Average error rate is: " << error << std::endl;
        std::cout << "Normalized error rate is: " << error/maxDistance << std::endl;
    }

    // Adjust the iteration count in performance metrics to reflect only the timed iterations
    int timedIterations = iters - 1;
    
    bh->printPerformanceMetrics();
    
    // Export final performance data
    std::ofstream perfOutput("performance_results.csv");
    if (perfOutput.is_open()) {
        perfOutput << "bodies,iterations,total_kernel_time_ms,avg_kernel_time_ms,total_execution_time_ms,avg_execution_time_ms,total_flops,kernel_gflops,effective_gflops" << std::endl;
        double kernelGflops = bh->getTotalFlops() / (bh->getTotalKernelTime() * 1e6);
        double effectiveGflops = bh->getTotalFlops() / (bh->getTotalExecutionTime() * 1e6);
        
        perfOutput << nBodies << ","
                  << timedIterations << ","
                  << bh->getTotalKernelTime() << ","
                  << bh->getAverageKernelTime() << ","
                  << bh->getTotalExecutionTime() << ","
                  << bh->getAverageExecutionTime() << ","
                  << bh->getTotalFlops() << ","
                  << kernelGflops << ","
                  << effectiveGflops << std::endl;
        
        perfOutput.close();
        std::cout << "Performance data written to performance_results.csv" << std::endl;
    } else {
        std::cerr << "ERROR: Could not open performance_results.csv for writing" << std::endl;
    }

    video.release();
    delete bh;
    return 0;
}
